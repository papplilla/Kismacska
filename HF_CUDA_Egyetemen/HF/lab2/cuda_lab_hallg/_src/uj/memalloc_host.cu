
#include <hip/hip_runtime.h>
#include "memory.h"

#define USE_CUDA_MEMALLOC 0
#define USE_WC 1

void memalloc_host(int imgWidthF, int imgHeightF, unsigned char **input, 
	               int imgWidth, int imgHeight, unsigned char **output)
{
	hipError_t cuda_ret;

#if USE_CUDA_MEMALLOC == 0
	*input   = (unsigned char *)malloc(3*imgWidthF*imgHeightF*sizeof(unsigned char));
	*output = (unsigned char *)malloc(3*imgWidth*imgHeight*sizeof(unsigned char));
#else
	#if USE_WC == 0
		cuda_ret = cudaHostAlloc(input, 3*imgWidthF*imgHeightF*sizeof(unsigned char), cudaHostAllocDefault);
	#else
		cuda_ret = cudaHostAlloc(input, 3*imgWidthF*imgHeightF*sizeof(unsigned char), cudaHostAllocWriteCombined);
	#endif
	cuda_ret = cudaHostAlloc(output, 3*imgWidth*imgHeight*sizeof(unsigned char), cudaHostAllocDefault);

#endif
}

void memfree_host(unsigned char **input, unsigned char **output)
{
	hipError_t cuda_ret;

#if USE_CUDA_MEMALLOC == 0
	free(*input);
	free(*output);
#else
	cuda_ret = cudaFree(input);
	cuda_ret = cudaFree(output);
#endif
}