#include "hip/hip_runtime.h"
#include <stdio.h>   
#include "time.h"
#include "omp.h"

#include "emmintrin.h"
#include "nmmintrin.h"

#include "defs.h"
#include "func.h"

#include <hip/hip_runtime.h>

__constant__ int filter_laplace[5][5] = { -1, -1, -1, -1, -1,
										 -1, -1, -1, -1, -1,
										 -1, -1, 24, -1, -1,
										 -1, -1, -1, -1, -1,
										 -1, -1, -1, -1, -1 };

__constant__ float filter_laplace_f[5][5] = { -1.0f, -1.0f, -1.0f, -1.0f, -1.0f,
											 -1.0f, -1.0f, -1.0f, -1.0f, -1.0f,
											 -1.0f, -1.0f, 24.0f, -1.0f, -1.0f,
											 -1.0f, -1.0f, -1.0f, -1.0f, -1.0f,
											 -1.0f, -1.0f, -1.0f, -1.0f, -1.0f };


// Glob�lis mem�ri�t haszn�l� (trivi�lis) megold�s
__global__ void kernel_conv_global(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int rd_base = (row * imgWidthF  +col)*3;
	int wr_base = (row*imgWidth+col) * 3;

	short fval[3] = { 0,0,0 };
	int rd_offset = 0;
	for (int fy = 0; fy<FILTER_H; fy++)
	{
		for (int fx = 0; fx<FILTER_W; fx++)									
		{
			for (int rgba = 0; rgba<3; rgba++)									// szinek
			{
				int pixel_src = rd_base + rd_offset;
				fval[rgba] = fval[rgba] + filter_laplace[fy][fx] * (short)(*(gInput + pixel_src + rgba));
				
			}
			rd_offset = rd_offset + 3;
		}
		rd_offset = rd_offset - 5 * 3 + imgWidthF * 3;

	}

	for (int rgba = 0; rgba<3; rgba++)
	{
		if (fval[rgba]>255)
			fval[rgba] = 255;
		else if (fval[rgba]<0)  
			fval[rgba] = 0;
	}

	for (int rgba = 0; rgba<3; rgba++)
	{
		*(gOutput + wr_base + rgba) = (unsigned char)(fval[rgba]);
	}



}

// Shared mem�ri�t haszn�l� megold�s (1.)
// Shared mem�ria adatt�pus: unsigned char
// Sz�m�t�s adatt�pusa: integer
__global__ void kernel_conv_sh_uchar_int(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  

  int th1D = threadIdx.y * blockDim.x + threadIdx.x;		// sz�l sz�ma
  int rx = blockIdx.x * blockDim.x;
  int ry = blockIdx.y * blockDim.y;

  int base = (ry*imgWidthF + rx) * 3;

  int wr_x, int wr_y;
  

  __shared__ unsigned char tomb[20][60];

 
	 
	// bet�lt�s
	  if (th1D < 240)
	  {
		  for (int i = 0; i < 5; i++)
		  {
			  wr_y = 4 * i + th1D / 60;
			  wr_x = th1D % 60;
			  tomb[wr_y][wr_x] = *(gInput + base + wr_y*imgWidthF + wr_x);
		  }
	  }

	  // sz�lak szinkroniz�l�sa
	  __syncthreads();


	  // konvol�ci�
	  //int rd_base = (row * imgWidthF + col) * 3;
	  int wr_base = (row*imgWidth + col) * 3;

	  short fval[3] = { 0,0,0 };
	  int rd_offset = 0;

	  for (int fy = 0; fy<FILTER_H; fy++)
	  {
		  for (int fx = 0; fx<FILTER_W; fx++)
		  {
			  for (int rgba = 0; rgba<3; rgba++)									// szinek
			  {
				  //int pixel_src = rd_base + rd_offset;
				  //fval[rgba] = fval[rgba] + filter_laplace[fy][fx] * (short)(*(gInput + pixel_src + rgba));

				  //fval[rgba] = fval[rgba] + filter_laplace[fy][fx] * (short)(*(tomb + pixel_src + rgba));
				  fval[rgba] = fval[rgba] + tomb[threadIdx.y + fy][threadIdx.x*3 + rgba + fx] * filter_laplace[fy][fx];
			  }
			  rd_offset = rd_offset + 3;
		  }
		  rd_offset = rd_offset - 5 * 3 + imgWidthF * 3;

	  }

	  for (int rgba = 0; rgba<3; rgba++)
	  {
		  if (fval[rgba]>255)
			  fval[rgba] = 255;
		  else if (fval[rgba]<0)
			  fval[rgba] = 0;
	  }

	  for (int rgba = 0; rgba<3; rgba++)
	  {
		  *(gOutput + wr_base + rgba) = (unsigned char)(fval[rgba]);
	  }

}


// Shared mem�ri�t haszn�l� megold�s (2.)
// Shared mem�ria adatt�pus: unsigned char
// Sz�m�t�s adatt�pusa: float
// A t�lt�s t�mb indexel�s helyett pointer + offset megold�ssal
__global__ void kernel_conv_sh_uchar_float(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

}

// Shared mem�ri�t haszn�l� megold�s (3.)
// Shared mem�ria adatt�pus: float
// Sz�m�t�s adatt�pusa: float
__global__ void kernel_conv_sh_float_float(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;






}

// Ugyanaz mint az el�bb, m�dos�tott blokk m�retekkel (32x8) az olvas�si shared memory bank konfliktus elker�l�s�hez
__global__ void kernel_conv_sh_float_float_nbc(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;






}



void cudaMain(int imgHeight, int imgWidth, int imgHeightF, int imgWidthF,
	          unsigned char *imgSrc, unsigned char *imgDst)

{
    double s0, e0;
    double d0;

	unsigned char *gInput, *gOutput;
	// GPU global memory foglal�s a bemeneti (kiterjesztett) k�pnek
    int size_in  = imgWidthF*imgHeightF*sizeof(unsigned char) * 3;
	hipMalloc((void**)&gInput, size_in);
	// GPU global memory foglal�s a kimeneti (nem kiterjesztett) k�pnek
	int size_out = imgWidth*imgHeight*sizeof(unsigned char) * 3;
	hipMalloc((void**)&gOutput, size_out);
    
	// 16x16 sz�las thread block
	dim3 thrBlock(16, 16);
    dim3 thrGrid(imgWidth/16, imgHeight/16);

	// 32x8 sz�las thread block
	dim3 thrBlock2(32, 8);
	dim3 thrGrid2(imgWidth / 32, imgHeight / 8);


	// bemeneti k�p m�sol�sa host --> GPU
	hipMemcpy(gInput, imgSrc, size_in, hipMemcpyHostToDevice); 

	// L1/Shared memory konfigur�ci�: sok cache
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	
	// L1/Shared memory konfigur�ci�: sok cache
	//hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	
	s0 = time_measure(1);
	for (int i = 0; i < KERNEL_RUNS; i++)
	{
		//kernel_conv_global << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		kernel_conv_sh_uchar_int << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		//kernel_conv_sh_uchar_float << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		//kernel_conv_sh_float_float << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		//kernel_conv_sh_float_float_nbc << <thrGrid2, thrBlock2 >> >(gInput, gOutput, imgWidth, imgWidthF);
	}
	hipDeviceSynchronize();
	e0 = time_measure(2);

	// Kimenet m�sol�s: GPU --> host
    hipMemcpy(imgDst, gOutput, size_out, hipMemcpyDeviceToHost);
	
	// GPU mem�ri�k felszabad�t�sa
    hipFree(gInput); hipFree(gOutput);

	// Reset (profiler miatt)
	hipDeviceReset();

    d0 = (double)(e0-s0)/(CLOCKS_PER_SEC*KERNEL_RUNS);
	double mpixel = (imgWidth*imgHeight / d0) / 1000000;
    printf("CUDA single kernel time: %4.4f\n", d0);
	printf("CUDA Mpixel/s: %4.4f\n", mpixel);
}