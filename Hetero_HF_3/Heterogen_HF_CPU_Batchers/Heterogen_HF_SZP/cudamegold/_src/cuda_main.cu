#include "hip/hip_runtime.h"
#include <stdio.h>   
#include "time.h"
#include "omp.h"

#include "emmintrin.h"
#include "nmmintrin.h"

#include "defs.h"
#include "func.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <hip/device_functions.h>


__constant__ int filter_laplace[5][5] = {-1, -1, -1, -1, -1,
                                         -1, -1, -1, -1, -1,
                                         -1, -1, 24, -1, -1,
                                         -1, -1, -1, -1, -1,
                                         -1, -1, -1, -1, -1};

__constant__ float filter_laplace_f[5][5] = {-1.0f, -1.0f, -1.0f, -1.0f, -1.0f,
                                             -1.0f, -1.0f, -1.0f, -1.0f, -1.0f,
                                             -1.0f, -1.0f, 24.0f, -1.0f, -1.0f,
                                             -1.0f, -1.0f, -1.0f, -1.0f, -1.0f,
                                             -1.0f, -1.0f, -1.0f, -1.0f, -1.0f};


// Glob�lis mem�ri�t haszn�l� (trivi�lis) megold�s
__global__ void kernel_conv_global(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  
  int out_pix = (row*imgWidth+col)*3;
  int pix_data[3] = {0, 0, 0};

  int in_pix = (row*imgWidthF + col)*3;
  #pragma unroll 5
  for (int fy=0; fy<5; fy++)
  {
      #pragma unroll 5
      for (int fx=0; fx<5; fx++)
      {
          #pragma unroll
          for (int rgba=0; rgba<3; rgba++)
          {
              pix_data[rgba] = pix_data[rgba] + (gInput[in_pix+rgba]*filter_laplace[fy][fx]);
          }
          in_pix=in_pix+3;
      }
      in_pix = in_pix + 3*imgWidthF - 5*1*3;
  }

  for (int rgba=0; rgba<3; rgba++)
  {
      pix_data[rgba] = min(pix_data[rgba], 255);
      pix_data[rgba] = max(pix_data[rgba],   0);
      gOutput[out_pix+rgba] = (unsigned char)(pix_data[rgba]);
  }
}

// Shared mem�ri�t haszn�l� megold�s (1.)
// Shared mem�ria adatt�pus: unsigned char
// Sz�m�t�s adatt�pusa: integer
__global__ void kernel_conv_sh_uchar_int(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  
  int out_pix = (row*imgWidth+col)*3;
  int pix_data[3] = {0, 0, 0};

  __shared__ unsigned char in_shmem[20][20][3];

  int thid_1d = threadIdx.y*blockDim.x + threadIdx.x;
  int ld_rgba = thid_1d%3;
  int ld_col  = (thid_1d/3)%20;
  int ld_row  = thid_1d / 60;
  int ld_base = (blockIdx.y * blockDim.y) * 3 * imgWidthF + (blockIdx.x * blockDim.x) * 3 + (ld_row * 3 * imgWidthF);
  if (thid_1d<(3*20*4))
  {
	  #pragma unroll
	  for (int ld=0; ld<5; ld++)
	  {
		  in_shmem[ld_row + ld * 4][ld_col][ld_rgba] = gInput[ld_base + (thid_1d % 60)];
		  ld_base = ld_base + imgWidthF*3*4;
	  }
  }
__syncthreads();

  #pragma unroll 5
  for (int fy=0; fy<5; fy++)
  {
      #pragma unroll 5
      for (int fx=0; fx<5; fx++)
      {
		  int coeff =  filter_laplace[fy][fx];
          #pragma unroll 3
          for (int rgba=0; rgba<3; rgba++)
          {
			  int pix = (int)(in_shmem[threadIdx.y+fy][threadIdx.x+fx][rgba]);
              pix_data[rgba] = pix_data[rgba] + (coeff * pix);
          }
      }
  }

  for (int rgba=0; rgba<3; rgba++)
  {
      pix_data[rgba] = min(pix_data[rgba], 255);
      pix_data[rgba] = max(pix_data[rgba],   0);
	  gOutput[out_pix + rgba] = (unsigned char)(pix_data[rgba]);
  }
}

// Shared mem�ri�t haszn�l� megold�s (2.)
// Shared mem�ria adatt�pus: unsigned char
// Sz�m�t�s adatt�pusa: float
// A t�lt�s t�mb indexel�s helyett pointer + offset megold�ssal
__global__ void kernel_conv_sh_uchar_float(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int out_pix = (row*imgWidth + col) * 3;
	float pix_data[3] = { 0.0f, 0.0f, 0.0f };

	__shared__ unsigned char in_shmem[20][20][3];

	int thid_1d = threadIdx.y*blockDim.x + threadIdx.x;
	int ld_row = thid_1d / 60;
	int ld_addr = (blockIdx.y * blockDim.y) * 3 * imgWidthF + (blockIdx.x * blockDim.x) * 3 + (ld_row * 3 * imgWidthF) + (thid_1d % 60);
	int smem_addr = thid_1d;
	unsigned char *smem_base_addr = &in_shmem[0][0][0];
	if (thid_1d<3 * 20 * 4)
	{
#pragma unroll
		for (int ld = 0; ld<5; ld++)
		{
			
			*(smem_base_addr + smem_addr) = gInput[ld_addr];
			ld_addr = ld_addr + imgWidthF * 3 * 4;
			smem_addr = smem_addr + 4 * 60;
		}
	}
__syncthreads();

#pragma unroll 5
	for (int fy = 0; fy<5; fy++)
	{
#pragma unroll 5
		for (int fx = 0; fx<5; fx++)
		{
			float coeff = filter_laplace_f[fy][fx];
#pragma unroll 3
			for (int rgba = 0; rgba<3; rgba++)
			{
				float pix = (float)(in_shmem[threadIdx.y + fy][threadIdx.x + fx][rgba]);
				pix_data[rgba] = pix_data[rgba] + (coeff * pix);
			}
		}
	}

	for (int rgba = 0; rgba<3; rgba++)
	{
		pix_data[rgba] = min(pix_data[rgba], 255.0f);
		pix_data[rgba] = max(pix_data[rgba], 0.0f);
		gOutput[out_pix + rgba] = (unsigned char)(pix_data[rgba]);
	}
}

// Shared mem�ri�t haszn�l� megold�s (3.)
// Shared mem�ria adatt�pus: float
// Sz�m�t�s adatt�pusa: float
__global__ void kernel_conv_sh_float_float(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int out_pix = (row*imgWidth + col) * 3;
	float pix_data[3] = { 0.0f, 0.0f, 0.0f };

	__shared__ float in_shmem[20][20][3];

	int thid_1d = threadIdx.y*blockDim.x + threadIdx.x;
	int ld_rgba = thid_1d % 3;
	int ld_col = (thid_1d / 3) % 20;
	int ld_row = thid_1d / 60;
	int ld_base = (blockIdx.y * blockDim.y) * 3 * imgWidthF + (blockIdx.x * blockDim.x) * 3 + (ld_row * 3 * imgWidthF);
	if (thid_1d<3 * 20 * 4)
	{
#pragma unroll
		for (int ld = 0; ld<5; ld++)
		{
			in_shmem[ld_row + ld * 4][ld_col][ld_rgba] = (float)(gInput[ld_base + (thid_1d % 60)]);
			ld_base = ld_base + imgWidthF * 3 * 4;
		}
	}
	__syncthreads();

#pragma unroll 5
	for (int fy = 0; fy<5; fy++)
	{
#pragma unroll 5
		for (int fx = 0; fx<5; fx++)
		{
			float coeff = filter_laplace_f[fy][fx];
#pragma unroll
			for (int rgba = 0; rgba<3; rgba++)
			{
				float pix = in_shmem[threadIdx.y + fy][threadIdx.x + fx][rgba];
				pix_data[rgba] = pix_data[rgba] + (coeff * pix);
			}
		}
	}

	for (int rgba = 0; rgba<3; rgba++)
	{
		pix_data[rgba] = min(pix_data[rgba], 255.0f);
		pix_data[rgba] = max(pix_data[rgba], 0.0f);
		gOutput[out_pix + rgba] = (unsigned char)(pix_data[rgba]);
	}
}

// Ugyanaz mint az el�bb, m�dos�tott blokk m�retekkel (32x8) az olvas�si shared memory bank konfliktus elker�l�s�hez
__global__ void kernel_conv_sh_float_float_nbc(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int out_pix = (row*imgWidth + col) * 3;
	float pix_data[3] = { 0.0f, 0.0f, 0.0f };

	__shared__ float in_shmem[8+4][32+4][3];

	int thid_1d = threadIdx.y*blockDim.x + threadIdx.x;
	int ld_rgba = thid_1d % 3;
	int ld_col = (thid_1d / 3) % (32+4);
	int ld_row = thid_1d / (3*(32+4));
	int ld_base = (blockIdx.y * blockDim.y) * 3 * imgWidthF + (blockIdx.x * blockDim.x) * 3 + (ld_row * 3 * imgWidthF);
	if (thid_1d<(2*3*(32+4)))
	{
#pragma unroll
		for (int ld = 0; ld<6; ld++)
		{
			in_shmem[ld_row + ld * 2][ld_col][ld_rgba] = (float)(gInput[ld_base + (thid_1d % (3 * (32 + 4)))]);
			ld_base = ld_base + imgWidthF * 3 * 2;
		}
	}
	__syncthreads();

#pragma unroll 5
	for (int fy = 0; fy<5; fy++)
	{
#pragma unroll 5
		for (int fx = 0; fx<5; fx++)
		{
			float coeff = filter_laplace_f[fy][fx];
#pragma unroll
			for (int rgba = 0; rgba<3; rgba++)
			{
				float pix = in_shmem[threadIdx.y + fy][threadIdx.x + fx][rgba];
				pix_data[rgba] = pix_data[rgba] + (coeff * pix);
			}
		}
	}

	for (int rgba = 0; rgba<3; rgba++)
	{
		pix_data[rgba] = min(pix_data[rgba], 255.0f);
		pix_data[rgba] = max(pix_data[rgba], 0.0f);
		gOutput[out_pix + rgba] = (unsigned char)(pix_data[rgba]);
	}
}



void cudaMain(int imgHeight, int imgWidth, int imgHeightF, int imgWidthF,
			  int imgFOfssetH, int imgFOfssetW,
			  unsigned char *imgSrc, unsigned char *imgDst)

{
    double s0, e0;
    double d0;


    int size_in  = imgWidthF*imgHeightF*sizeof(unsigned char) * 3;
	int size_out = imgWidth*imgHeight*sizeof(unsigned char) * 3;

    unsigned char *gInput, *gOutput;
	hipMalloc((void**)&gInput, size_in);
	hipMalloc((void**)&gOutput, size_out);

	dim3 thrBlock(16, 16);
    dim3 thrGrid(imgWidth/16, imgHeight/16);

	dim3 thrBlock2(32, 8);
	dim3 thrGrid2(imgWidth / 32, imgHeight / 8);

	hipMemcpy(gInput, imgSrc, size_in, hipMemcpyHostToDevice); 

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	s0 = time_measure(1);
	for (int i = 0; i < KERNEL_RUNS; i++)
	{
		//kernel_conv_global << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		//kernel_conv_sh_uchar_int << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		//kernel_conv_sh_uchar_float << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		//kernel_conv_sh_float_float << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		kernel_conv_sh_float_float_nbc << <thrGrid2, thrBlock2 >> >(gInput, gOutput, imgWidth, imgWidthF);
	}
	hipDeviceSynchronize();
	e0 = time_measure(2);

    hipMemcpy(imgDst, gOutput, size_out, hipMemcpyDeviceToHost);
	
    hipFree(gInput); hipFree(gOutput);

	hipDeviceReset();

    d0 = (double)(e0-s0)/(CLOCKS_PER_SEC*KERNEL_RUNS);
	double mpixel = (imgWidth*imgHeight / d0) / 1000000;
    printf("CUDA single kernel time: %4.4f\n", d0);
	printf("CUDA Mpixel/s: %4.4f\n", mpixel);
}